#include "hip/hip_runtime.h"
/* -*- C++ -*- */
/*    This file is part of CUDA_tests
      Copyright (C) 2016  Julien Thevenon ( julien_thevenon at yahoo.fr )

      This program is free software: you can redistribute it and/or modify
      it under the terms of the GNU General Public License as published by
      the Free Software Foundation, either version 3 of the License, or
      (at your option) any later version.

      This program is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
      GNU General Public License for more details.

      You should have received a copy of the GNU General Public License
      along with this program.  If not, see <http://www.gnu.org/licenses/>
*/
#include <iostream>
#include <iomanip>
#include <cinttypes>
#include <cstring>

#include "my_cuda.h"

COMMON_KERNEL_ATTRIBUTES
void common_kernel(MY_CUDA_PARAMS_DECL uint32_t * p_int_ptr, uint32_t * p_nipples_ptr)
{
  int l_shift = threadIdx.x << 2;
  uint32_t l_mask = ((uint32_t)0xF) << l_shift;
  p_nipples_ptr[threadIdx.x] = (*p_int_ptr & l_mask) >> l_shift;
}
__global__
void cuda_kernel(MY_CUDA_PARAMS_DECL uint32_t * p_int_ptr, uint32_t * p_nipples_ptr)
{
  common_kernel(MY_CUDA_PARAMS_INST p_int_ptr, p_nipples_ptr);
}


int main(void)
{
  uint32_t l_int = 0x87654321;
  uint32_t l_nipples[8] = {0, 0, 0, 0, 0, 0, 0, 0};

  uint32_t * l_int_ptr;
  uint32_t * l_nipples_ptr;

  hipMalloc(&l_int_ptr, sizeof(uint32_t));
  hipMalloc(&l_nipples_ptr, sizeof(uint32_t) * 8);
  hipMemcpy(l_int_ptr, &l_int, sizeof(uint32_t), hipMemcpyHostToDevice);
  hipMemcpy(l_nipples_ptr, &l_nipples[0], sizeof(uint32_t) * 8, hipMemcpyHostToDevice);

  dim3 dimBlock(8, 1);
  dim3 dimGrid( 1, 1);
  launch_kernels(cuda_kernel,dimGrid, dimBlock,l_int_ptr, l_nipples_ptr);

  hipMemcpy(l_nipples, l_nipples_ptr, sizeof(uint32_t) * 8, hipMemcpyDeviceToHost);
  hipFree(l_nipples_ptr);
  hipFree(l_int_ptr);

  std::cout << std::hex << "0x" << l_int << std::dec << std::endl ;
  for(unsigned int l_index = 0; l_index < 8; ++l_index)
    {
      std::cout << "Nipple[" << l_index << "] = 0x" << std::hex <<  l_nipples[l_index] << std::dec << std::endl ;
    } 
  return EXIT_SUCCESS;
}
// EOF
